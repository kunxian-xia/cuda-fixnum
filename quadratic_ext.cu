#pragma once
#include <hip/hip_runtime.h>


// #include "fixnum/word_fixnum.cu"

template < typename fixnum, typename monty > 
class quad_ext_element {
public:
    typedef fixnum modnum;
    modnum a0;
    modnum a1;

    __device__ quad_ext_element() { }

    __device__ static void to_modnum(monty mod, quad_ext_element &z) {
        modnum t0, t1;
        mod.to_modnum(t0, z.a0);
        mod.to_modnum(t1, z.a1);

        z.a0 = t0; z.a1 = t1;
    }

    __device__ static void from_modnum(monty mod, quad_ext_element &z) {
        fixnum t0, t1;
        mod.from_modnum(t0, z.a0);
        mod.from_modnum(t1, z.a1);
        z.a0 = t0; z.a1 = t1;
    }
    /*
    __device__ quad_ext_element(monty mod, fixnum z0, fixnum z1) {
        mod.to_modnum(a0, z0);
        mod.to_modnum(a1, z1);
    }
    */
    // __device__ quad_ext_element(modnum z0, modnum z1) : a0(z0), a1(z1) {  printf("cons"); }
};

template < typename fixnum, typename monty >
class quad_ext {
public:
    typedef fixnum modnum;
    monty mod;
    modnum alpha; 

    typedef quad_ext_element<fixnum, monty> quad_ext_element;

    __device__ quad_ext(fixnum modulus, modnum _alpha) : mod(modulus), alpha(_alpha) {}

    __device__ void add(quad_ext_element &z, quad_ext_element &x, quad_ext_element &y) {
        modnum t0, t1;
        mod.add(t0, x.a0, y.a0);
        mod.add(t1, x.a1, y.a1);
        z.a0 = t0; z.a1 = t1;
    }

    __device__ void mul(quad_ext_element &z, quad_ext_element x, quad_ext_element y) {
        modnum t0, t1, t2;

        // c0 = a0*b0 + 13*a1*b1
        mod.mul(t0, x.a0, y.a0);
        mod.mul(t1, x.a1, y.a1);
        mod.mul(t2, alpha, t1);
        mod.add(z.a0, t0, t2);

        // c1 = a0*b1 + a1*b0
        mod.mul(t0, x.a0, y.a1);
        mod.mul(t1, x.a1, y.a0);
        mod.add(z.a1, t0, t1);
    }
};
