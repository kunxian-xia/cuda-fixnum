#include "hip/hip_runtime.h"

#include <cstdio>
// #include <cstring>
// #include <cassert>
#include <vector>

#include "fixnum/warp_fixnum.cu"
#include "array/fixnum_array.h"
#include "functions/modexp.cu"
#include "functions/multi_modexp.cu"
#include "modnum/modnum_monty_redc.cu"
#include "modnum/modnum_monty_cios.cu"
#include "quadratic_ext.cu"
#include "cubic_ext.cu"

const unsigned int bytes_per_elem = 128;
const unsigned int io_bytes_per_elem = 96;

using namespace std;
using namespace cuFIXNUM;

// 1. read_mnt4_fq2
// 2. allocate cuda memory for input vectors
// 3. dispatch the work to input
// 4. get output back 
// 5. write_mnt4_fq2

// template< typename fixnum >
// struct quad_mul {
//     typedef modnum_monty_redc<fixnum> modnum_redc;
//     typedef quad_ext_element<fixnum> quad;
//     typedef quad_ext<fixnum, modnum_redc> quad_ext;
//     __device__ void operator()(fixnum alpha, fixnum modulus, quad a, quad b, quad &r) {
//         quad_ext ext(modulus, alpha);

//         ext.to_modnum(a);
//         ext.to_modnum(b);
//         ext.mul(r, a, b);

//         ext.from_modnum(r);
//     }
// };

template< typename fixnum >
struct cubic_mul {
    typedef modnum_monty_redc<fixnum> modnum_redc;
    typedef cubic_ext_element<fixnum> cubic;
    typedef cubic_ext<fixnum, modnum_redc> cubic_ext;
    __device__ void operator()(fixnum alpha, fixnum modulus, fixnum rr, cubic a, cubic b, cubic &r) {
        cubic_ext ext(modulus, alpha);

        // ext.to_modnum(a);
        // ext.to_modnum(b);
        cubic t;
        ext.mul(t, a, b);

        modnum_redc mod(modulus);
        fixnum t0, t1, t2;
        mod.mul(r.a0, t.a0, rr);
        mod.mul(r.a1, t.a1, rr);
        mod.mul(r.a2, t.a2, rr);
        // r.a0 = t0; r.a1 = t1; r.a2 = t2;
        // cubic tmp;
        // tmp.a0 = rr; tmp.a1 = rr; tmp.a2 = rr;
        // ext.mul(r, t, tmp);
        // ext.from_modnum(r);
    }
};

template <typename fixnum, template<typename> class Func > 
__global__ void dispatch(int nelts, fixnum *alpha, fixnum *modulus, fixnum *rr, cubic_ext_element<fixnum> *a, 
    cubic_ext_element<fixnum> *b, cubic_ext_element<fixnum> *c)
{
    int blk_tid_offset = blockDim.x * blockIdx.x;
    int tid_in_blk = threadIdx.x;
    int idx = (blk_tid_offset + tid_in_blk) / fixnum::SLOT_WIDTH;

    if (idx < nelts) {
        Func<fixnum> fn;
        // TODO: This offset calculation is entwined with fixnum layout and so
        // belongs somewhere else.
        int off = idx * fixnum::layout::WIDTH + fixnum::layout::laneIdx();
        
        // TODO: This is hiding a sin against memory aliasing / management /
        // type-safety.
        fn(alpha[off], modulus[off], rr[off], a[off], b[off], c[off]);
    }
}

template <typename fixnum>
void mnt_fq3_to_cubic_element(uint8_t *fq3, cubic_ext_element<fixnum> *ele) {
  uint8_t* data = reinterpret_cast<uint8_t*>(ele);
  int bytes = fixnum::BYTES;
  int word_size = sizeof(fixnum);

  for (int i = 0; i < bytes/word_size; i++) {
      for (int j = 0; j < word_size; j++) {
          data[3*i*word_size+j] = fq3[i*word_size+j];
          data[(3*i+1)*word_size+j] = fq3[bytes_per_elem+i*word_size+j];
          data[(3*i+2)*word_size+j] = fq3[2*bytes_per_elem+i*word_size+j];
      }
  }
}

template <typename fixnum >
uint8_t* cubic_element_to_mnt_fq3(cubic_ext_element<fixnum> *ele) {
    uint8_t *r = new uint8_t[fixnum::BYTES*3];
    uint8_t *data = reinterpret_cast<uint8_t*>(ele);
    int word_size = sizeof(fixnum);

    for (unsigned int i = 0; i < bytes_per_elem/word_size; i++) {
        for (int j = 0; j < word_size; j++) {
            r[i*word_size+j] = data[3*i*word_size+j];
            r[bytes_per_elem+i*word_size+j] = data[(3*i+1)*word_size+j];
            r[2*bytes_per_elem+i*word_size+j] = data[(3*i+2)*word_size+j];
        }
    }
    return r;
}

template <int fn_bytes, typename word_fixnum, template <typename> class Func>
std::vector<uint8_t*> compute_product(std::vector<uint8_t*> a, std::vector<uint8_t*> b, uint8_t *modulus, uint8_t *alpha, uint8_t *rr) {
    typedef warp_fixnum<fn_bytes, word_fixnum> fixnum;
    typedef cubic_ext_element<fixnum> cubic;
    typedef cubic_ext<fixnum, modnum_monty_redc<fixnum>> ext;

    int n = a.size();
    cubic *inputs_a, *inputs_b, *output;
    cuda_malloc_managed((void**)&inputs_a, fn_bytes*3*n);
    cuda_malloc_managed((void**)&inputs_b, fn_bytes*3*n);
    cuda_malloc_managed((void**)&output, fn_bytes*3*n);
  
    for (int i = 0; i < n; i++) {
        mnt_fq3_to_cubic_element<fixnum>(a[i], &inputs_a[i*fn_bytes*3/sizeof(cubic)]);
        mnt_fq3_to_cubic_element<fixnum>(b[i], &inputs_b[i*fn_bytes*3/sizeof(cubic)]);
    }

    fixnum *inputs_mod, *inputs_alpha, *inputs_rr;
    cuda_malloc_managed(&inputs_mod, fn_bytes*n);
    cuda_malloc_managed(&inputs_alpha, fn_bytes*n);
    cuda_malloc_managed(&inputs_rr, fn_bytes*n);
    for (int i = 0; i < n; i++) {
        fixnum::from_bytes(reinterpret_cast<uint8_t*>(
            &inputs_mod[i*fn_bytes/sizeof(fixnum)]), modulus, fixnum::BYTES);
        fixnum::from_bytes(reinterpret_cast<uint8_t*>(
            &inputs_alpha[i*fn_bytes/sizeof(fixnum)]), alpha, fixnum::BYTES);
        fixnum::from_bytes(reinterpret_cast<uint8_t*>(
            &inputs_rr[i*fn_bytes/sizeof(fixnum)]), rr, fixnum::BYTES);
    }

    constexpr int BLOCK_SIZE = 192;
    constexpr int fixnums_per_block = BLOCK_SIZE / fixnum::SLOT_WIDTH;
    int nblocks = ceilquo(n, fixnums_per_block);
    hipStream_t stream;

    cuda_check(hipStreamCreate(&stream), "create stream");
    cuda_check(hipStreamSynchronize(stream), "stream sync");

    dispatch<fixnum, Func><<<nblocks, BLOCK_SIZE, 0, stream>>>(n, inputs_alpha, inputs_mod, inputs_rr, inputs_a, inputs_b, output);
    cuda_check(hipPeekAtLastError(), "kernel invocation/run");
    cuda_check(hipStreamSynchronize(stream), "stream sync");
    cuda_check(hipStreamDestroy(stream), "stream destroy");

    cuda_device_synchronize();

    std::vector<uint8_t *> ret;
    for (int i = 0; i < n; i++) {
        ret.emplace_back( cubic_element_to_mnt_fq3<fixnum>(&output[i*fn_bytes*3/sizeof(cubic)]) );
    }

    cuda_free(inputs_a);
    cuda_free(inputs_b);
    cuda_free(output);
    cuda_free(inputs_mod); cuda_free(inputs_alpha); cuda_free(inputs_rr);
    return ret;
}

uint8_t* read_mnt_fq3(FILE *inputs) {
  uint8_t* buf = (uint8_t*)calloc(bytes_per_elem*3, sizeof(uint8_t));
  fread((void*)(buf+bytes_per_elem-io_bytes_per_elem), io_bytes_per_elem*sizeof(uint8_t), 1, inputs);
  fread((void*)(buf+2*bytes_per_elem-io_bytes_per_elem), io_bytes_per_elem*sizeof(uint8_t), 1, inputs);
  fread((void*)(buf+3*bytes_per_elem-io_bytes_per_elem), io_bytes_per_elem*sizeof(uint8_t), 1, inputs);
  // fread((void*) buf, io_bytes_per_elem*sizeof(uint8_t), 1, inputs);
  // fread((void*) (buf+bytes_per_elem), io_bytes_per_elem*sizeof(uint8_t), 1, inputs);
  // fread((void*) (buf+2*bytes_per_elem), io_bytes_per_elem*sizeof(uint8_t), 1, inputs);
  return buf;
}

void write_mnt_fq3(uint8_t* fq3, FILE* outputs) {
  fwrite((void *) fq3, io_bytes_per_elem * sizeof(uint8_t), 1, outputs);
  fwrite((void *) (fq3+bytes_per_elem), io_bytes_per_elem * sizeof(uint8_t), 1, outputs);
  fwrite((void *) (fq3+2*bytes_per_elem), io_bytes_per_elem * sizeof(uint8_t), 1, outputs);
}

int main(int argc, char* argv[]) {
  if (argc < 4) {
      printf("usage: ./main compute inputs output\n");
      exit(1);
  }

  setbuf(stdout, NULL);

  // mnt4_q
  uint8_t mnt4_modulus[bytes_per_elem] = {1,128,94,36,222,99,144,94,159,17,221,44,82,84,157,227,240,37,196,154,113,16,136,99,164,84,114,118,233,204,90,104,56,126,83,203,165,13,15,184,157,5,24,242,118,231,23,177,157,247,90,161,217,36,209,153,141,237,160,232,37,185,253,7,115,216,151,108,249,232,183,94,237,175,143,91,80,151,249,183,173,205,226,238,34,144,34,16,17,196,146,45,198,196,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
  uint8_t rr[bytes_per_elem] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
  // mnt6_q
  uint8_t mnt6_modulus[bytes_per_elem] = {1,0,0,64,226,118,7,217,79,58,161,15,23,153,160,78,151,87,0,63,188,129,195,214,164,58,153,52,118,249,223,185,54,38,33,41,148,202,235,62,155,169,89,200,40,92,108,178,157,247,90,161,217,36,209,153,141,237,160,232,37,185,253,7,115,216,151,108,249,232,183,94,237,175,143,91,80,151,249,183,173,205,226,238,34,144,34,16,17,196,146,45,198,196,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};

  printf("%d\n", rr[96]);
  uint8_t alpha[bytes_per_elem] = {11};

  auto inputs = fopen(argv[2], "r");
  auto outputs = fopen(argv[3], "w");

  size_t n;

   while (true) {
    size_t elts_read = fread((void *) &n, sizeof(size_t), 1, inputs);
    if (elts_read == 0) { break; }
    printf("%lu\n", n);
    std::vector<uint8_t*> x0;
    for (size_t i = 0; i < n; ++i) {
      x0.emplace_back(read_mnt_fq3(inputs));
    }

    std::vector<uint8_t*> x1;
    for (size_t i = 0; i < n; ++i) {
      x1.emplace_back(read_mnt_fq3(inputs));
    }

    std::vector<uint8_t*> ret = compute_product<bytes_per_elem, u64_fixnum, cubic_mul>(x0, x1, mnt6_modulus, alpha, rr);
    for (size_t i = 0; i < n; ++i) {
        write_mnt_fq3(ret[i], outputs);
    }
    for (size_t i = 0; i < n;++i) {
        free(x0[i]);
        free(x1[i]);
        free(ret[i]);
    }
   }
   fclose(inputs);
   fclose(outputs);
}
