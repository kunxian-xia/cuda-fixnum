#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstring>
#include <cassert>
#include <vector>

#include "fixnum/warp_fixnum.cu"
#include "array/fixnum_array.h"
#include "functions/modexp.cu"
#include "functions/multi_modexp.cu"
#include "modnum/modnum_monty_redc.cu"
#include "modnum/modnum_monty_cios.cu"

const unsigned int bytes_per_elem = 128;
const unsigned int io_bytes_per_elem = 96;


using namespace std;
using namespace cuFIXNUM;

template< typename fixnum >
struct mul_and_convert {
  // redc may be worth trying over cios
  typedef modnum_monty_cios<fixnum> modnum;
  __device__ void operator()(fixnum &r, fixnum a, fixnum b, fixnum my_mod) {
      modnum mod = modnum(my_mod);

      fixnum sm;
      mod.mul(sm, a, b);

      fixnum s;
      mod.from_modnum(s, sm);

      r = s;
  }
};

template< typename fixnum >
struct qe2_mul_and_convert {
  typedef modnum_monty_cios<fixnum> modnum;
  __device__ void operator()(fixnum &r0, fixnum &r1, fixnum a0,
    fixnum a1, fixnum b0, fixnum b1, fixnum my_mod, fixnum alpha) {
      modnum mod = modnum(my_mod);

      fixnum alphaP;
      mod.to_modnum(alphaP, alpha); 

      fixnum c0; fixnum c1;
      fixnum t1, t2, t3, t4, t5;
      {
        mod.mul(t1, a0, b1);
        mod.mul(t2, a1, b0);
        mod.add(c1, t1, t2);
      }

      {
        mod.mul(t3, a0, b0);
        mod.mul(t4, a1, b1);
        mod.mul(t5, alphaP, t4);
        mod.add(c0, t3, t5);
      }
      
      fixnum s0; fixnum s1;
      mod.from_modnum(s0, c0);
      mod.from_modnum(s1, c1);
      r0 = s0;
      r1 = s1;
  }
};

template< int fn_bytes, typename fixnum_array >
void print_fixnum_array(fixnum_array* res, int nelts) {
    int lrl = fn_bytes*nelts;
    uint8_t local_results[lrl];
    int ret_nelts;
    for (int i = 0; i < lrl; i++) {
      local_results[i] = 0;
    }
    res->retrieve_all(local_results, fn_bytes*nelts, &ret_nelts);

    for (int i = 0; i < lrl; i++) {
      printf("%i ", local_results[i]);
    }
    printf("\n");
}

template< int fn_bytes, typename fixnum_array >
vector<uint8_t*> get_fixnum_array(fixnum_array* res, int nelts) {
    int lrl = fn_bytes*nelts;
    uint8_t local_results[lrl];
    int ret_nelts;
    for (int i = 0; i < lrl; i++) {
      local_results[i] = 0;
    }
    res->retrieve_all(local_results, fn_bytes*nelts, &ret_nelts);
    vector<uint8_t*> res_v;
    for (int n = 0; n < nelts; n++) {
      uint8_t* a = (uint8_t*)malloc(fn_bytes*sizeof(uint8_t));
      for (int i = 0; i < fn_bytes; i++) {
        a[i] = local_results[n*fn_bytes + i];
      }
      res_v.emplace_back(a);
    }
    return res_v;
}

struct pair
{
  uint8_t *a;
  uint8_t *b;
};

template< int fn_bytes, typename word_fixnum, template <typename> class Func >
std::vector<uint8_t*> compute_product(std::vector<uint8_t*> a0, std::vector<uint8_t*> a1, 
  std::vector<uint8_t*> b0, std::vector<uint8_t*> b1, uint8_t* input_m_base, uint8_t* alpha_base) {
    typedef warp_fixnum<fn_bytes, word_fixnum> fixnum;
    typedef fixnum_array<fixnum> fixnum_array;

    int nelts = a0.size();

    uint8_t *input_a0 = new uint8_t[fn_bytes * nelts];
    uint8_t *input_a1 = new uint8_t[fn_bytes * nelts];
    for (int i = 0; i < fn_bytes * nelts; ++i) {
      input_a0[i] = a0[i/fn_bytes][i%fn_bytes];
      input_a1[i] = a1[i/fn_bytes][i%fn_bytes];
    }

    uint8_t *input_b0 = new uint8_t[fn_bytes * nelts];
    uint8_t *input_b1 = new uint8_t[fn_bytes * nelts];
    for (int i = 0; i < fn_bytes * nelts; ++i) {
      input_b0[i] = b0[i/fn_bytes][i%fn_bytes];
      input_b1[i] = b1[i/fn_bytes][i%fn_bytes];
    }

    uint8_t *input_m = new uint8_t[fn_bytes * nelts];
    uint8_t *alpha = new uint8_t[fn_bytes * nelts];
    for (int i = 0; i < fn_bytes * nelts; ++i) {
      input_m[i] = input_m_base[i%fn_bytes];
      alpha[i] = alpha_base[i%fn_bytes];
    }

    // TODO reuse modulus as a constant instead of passing in nelts times
    fixnum_array *res0, *in_a0, *in_b0, *inM;
    fixnum_array *res1, *in_a1, *in_b1, *inA;
    in_a0 = fixnum_array::create(input_a0, fn_bytes * nelts, fn_bytes);
    in_a1 = fixnum_array::create(input_a1, fn_bytes * nelts, fn_bytes);
    in_b0 = fixnum_array::create(input_b0, fn_bytes * nelts, fn_bytes);
    in_b1 = fixnum_array::create(input_b1, fn_bytes * nelts, fn_bytes);
    inM = fixnum_array::create(input_m, fn_bytes * nelts, fn_bytes);
    inA = fixnum_array::create(alpha, fn_bytes * nelts, fn_bytes);
    res0 = fixnum_array::create(nelts);
    res1 = fixnum_array::create(nelts);

    fixnum_array::template map<Func>(res0, res1, in_a0, in_a1, in_b0, in_b1, inM, inA);

    vector<uint8_t*> v_res0 = get_fixnum_array<fn_bytes, fixnum_array>(res0, nelts);
    vector<uint8_t*> v_res1 = get_fixnum_array<fn_bytes, fixnum_array>(res1, nelts);

    //TODO to do stage 1 field arithmetic, instead of a map, do a reduce

    delete in_a0;
    delete in_a1;
    delete in_b0;
    delete in_b1;
    delete inM;
    delete inA;
    delete res0;
    delete res1;
    delete[] input_a0;
    delete[] input_a1;
    delete[] input_b0;
    delete[] input_b1;
    delete[] input_m;
    delete[] alpha;
    return v_res0;
}

uint8_t* read_mnt_fq(FILE* inputs) {
  uint8_t* buf = (uint8_t*)calloc(bytes_per_elem, sizeof(uint8_t));
  // the input is montgomery representation x * 2^768 whereas cuda-fixnum expects x * 2^1024 so we shift over by (1024-768)/8 bytes
  fread((void*)( buf + (bytes_per_elem - io_bytes_per_elem)), io_bytes_per_elem*sizeof(uint8_t), 1, inputs);
  return buf;
}

void write_mnt_fq(uint8_t* fq, FILE* outputs) {
  fwrite((void *) fq, io_bytes_per_elem * sizeof(uint8_t), 1, outputs);
}

int main(int argc, char* argv[]) {
  setbuf(stdout, NULL);

  // mnt4_q
  uint8_t mnt4_modulus[bytes_per_elem] = {1,128,94,36,222,99,144,94,159,17,221,44,82,84,157,227,240,37,196,154,113,16,136,99,164,84,114,118,233,204,90,104,56,126,83,203,165,13,15,184,157,5,24,242,118,231,23,177,157,247,90,161,217,36,209,153,141,237,160,232,37,185,253,7,115,216,151,108,249,232,183,94,237,175,143,91,80,151,249,183,173,205,226,238,34,144,34,16,17,196,146,45,198,196,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
  uint8_t alpha[bytes_per_elem] = {13};
  // mnt6_q
  uint8_t mnt6_modulus[bytes_per_elem] = {1,0,0,64,226,118,7,217,79,58,161,15,23,153,160,78,151,87,0,63,188,129,195,214,164,58,153,52,118,249,223,185,54,38,33,41,148,202,235,62,155,169,89,200,40,92,108,178,157,247,90,161,217,36,209,153,141,237,160,232,37,185,253,7,115,216,151,108,249,232,183,94,237,175,143,91,80,151,249,183,173,205,226,238,34,144,34,16,17,196,146,45,198,196,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};

  auto inputs = fopen(argv[2], "r");
  auto outputs = fopen(argv[3], "w");

  size_t n;

   while (true) {
    size_t elts_read = fread((void *) &n, sizeof(size_t), 1, inputs);
    if (elts_read == 0) { break; }

    std::vector<uint8_t*> x0c0;
    std::vector<uint8_t*> x0c1;
    for (size_t i = 0; i < n/2; ++i) {
      x0c0.emplace_back(read_mnt_fq(inputs));
      x0c1.emplace_back(read_mnt_fq(inputs));
    }

    std::vector<uint8_t*> x1c0;
    std::vector<uint8_t*> x1c1;
    for (size_t i = 0; i < n/2; ++i) {
      x1c0.emplace_back(read_mnt_fq(inputs));
      x1c1.emplace_back(read_mnt_fq(inputs));
    }

    std::vector<uint8_t*> res_x = compute_product<bytes_per_elem, u64_fixnum, qe2_mul_and_convert>(x0c0, x0c1, x1c0, x1c1, mnt4_modulus, alpha);

    for (size_t i = 0; i < n/2; ++i) {
      write_mnt_fq(res_x[i], outputs);
    }

    // std::vector<uint8_t*> y0;
    // for (size_t i = 0; i < n/2; ++i) {
    //   y0.emplace_back(read_mnt_fq(inputs));
    // }

    // std::vector<uint8_t*> y1;
    // for (size_t i = 0; i < n/2; ++i) {
    //   y1.emplace_back(read_mnt_fq(inputs));
    // }

    // std::vector<uint8_t*> res_y = compute_product<bytes_per_elem, u64_fixnum, mul_and_convert>(y0, y1, mnt6_modulus);

    // for (size_t i = 0; i < n/2; ++i) {
    //   write_mnt_fq(res_y[i], outputs);
    // }

    for (size_t i = 0; i < n/2; ++i) {
      free(x0c0[i]);
      free(x0c1[i]);
      free(x1c0[i]);
      free(x1c1[i]);
      free(res_x[i]);
      // free(res_y[i]);
    }

  }

  return 0;
}

